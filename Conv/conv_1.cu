#include "cudnn_cnn_infer.h"
#include <chrono>
#include <hipDNN.h>
#include <iostream>
#include <opencv2/dnn.hpp>
#include <opencv2/opencv.hpp>

// Check cudnn
#define checkCUDNN(expression)                                     \
    {                                                              \
        hipdnnStatus_t status = (expression);                       \
        if (status != HIPDNN_STATUS_SUCCESS) {                      \
            std::cerr << "Error on line " << __LINE__ << ": "      \
                      << hipdnnGetErrorString(status) << std::endl; \
            std::exit(EXIT_FAILURE);                               \
        }                                                          \
    }


// Load Image function
cv::Mat load_image(const char *image_path) {
    cv::Mat image = cv::imread(image_path, cv::IMREAD_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

// Save Image function
void save_image(const char *output_filename, float *buffer, int height,
                int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    // Make negative values zero.
    cv::threshold(output_image, output_image, 0, 0, cv::THRESH_TOZERO);
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);

    cv::imwrite(output_filename, output_image);
}

int main(int argc, char *argv[]) {
    // Init cudnn
    hipDeviceReset();

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    char *outputfile = (char *) "cudnn_out.png";
    // Check input image name
    if (argc < 2) {
        std::cout << "No file input" << std::endl;
        return 0;
    }
    //
    // Check if the filename is valid
    char *filename = argv[1];
    // std::cout << argv[1] << " ";
    // Load Image
    cv::Mat image;
    image = load_image(filename);
    // cv::Mat image = cv::dnn::blobFromImage(t, 1.0f, cv::Size(256,256), cv::Scalar(0,0,0));

    if (image.empty()) {
        std::cout << "File not exist" << std::endl;
        return 0;
    }
    auto timeStamp0 = std::chrono::high_resolution_clock::now();

    // Input Descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor))
            checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                                  /*format=*/HIPDNN_TENSOR_NHWC,
                                                  /*dataType=*/HIPDNN_DATA_FLOAT,
                                                  /*batch_size=*/1,
                                                  /*channels=*/3,
                                                  /*image_height=*/image.rows,
                                                  /*image_width=*/image.cols))

                    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor))
            checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                                  /*format=*/HIPDNN_TENSOR_NHWC,
                                                  /*dataType=*/HIPDNN_DATA_FLOAT,
                                                  /*batch_size=*/1,
                                                  /*channels=*/3,
                                                  /*image_height=*/image.rows,
                                                  /*image_width=*/image.cols))

                    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor))
            checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                                  /*dataType=*/HIPDNN_DATA_FLOAT,
                                                  /*format=*/HIPDNN_TENSOR_NCHW,
                                                  /*out_channels=*/3,
                                                  /*in_channels=*/3,
                                                  /*kernel_height=*/3,
                                                  /*kernel_width=*/3))

                    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                               /*pad_height=*/1,
                                               /*pad_width=*/1,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
    //  checkCUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
    //      cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
    //      output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
    //      /*memoryLimitInBytes=*/0, &convolution_algorithm));
    // convolution_algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
    // std::cout << "Cudnn Forward Algorithm Index :" << convolution_algorithm << std::endl;
    size_t workspace_bytes;

    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
            cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
            output_descriptor, convolution_algorithm, &workspace_bytes))
            // std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
            //           << std::endl;

            void *d_workspace;
    hipMalloc(&d_workspace, workspace_bytes);
    // std::cout << "allocate workspace" << std::endl;
    int batch_size;
    int channels;
    int height;
    int width;

    hipdnnGetConvolution2dForwardOutputDim(
            convolution_descriptor, input_descriptor, kernel_descriptor, &batch_size,
            &channels, &height, &width);

    int image_bytes = batch_size * channels * height * width * sizeof(float);

    float *d_input;
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

    float *d_output;
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // std::cout << "Height and width:" << height << " x " << width << std::endl;
    std::cout << height << " "
              << width << " ";
    // Mystery kernel
    const float kernel_template[3][3] = {{1, 1, 1},
                                         {1, -8, 1},
                                         {1, 1, 1}};
    //const float kernel_template[3][3] = {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}};

    float h_kernel[3][3][3][3];
    for (auto &kernel: h_kernel) {
        for (auto &channel: kernel) {
            for (int row = 0; row < 3; ++row) {
                for (int column = 0; column < 3; ++column) {
                    channel[row][column] = kernel_template[row][column];
                }
            }
        }
    }

    float *d_kernel;
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
    const float alpha = 1, beta = 0;

    // std::cout << "Start conv" << std::endl;
    auto timeStampA = std::chrono::high_resolution_clock::now();
    checkCUDNN(hipdnnConvolutionForward(
            cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel,
            convolution_descriptor, convolution_algorithm, d_workspace,
            workspace_bytes, &beta, output_descriptor, d_output))

            hipDeviceSynchronize();
    auto timeStampB = std::chrono::high_resolution_clock::now();
    auto *h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    auto timeStamp1 = std::chrono::high_resolution_clock::now();

    auto total_duration =
            std::chrono::duration_cast<std::chrono::milliseconds>(timeStamp1 - timeStamp0).count();

    auto conv_duration =
            std::chrono::duration_cast<std::chrono::milliseconds>(timeStampB - timeStampA).count();

    // Print result
    // std::cout << "Total process time: " << total_duration
    //           << std::endl;
    // std::cout << "Total convolution time: " << conv_duration
    //           << std::endl;
    // std::cout << "Save Output to " << outputfile << std::endl;
    save_image(outputfile, h_output, height, width);
    std::cout << conv_duration << "\n";
    // Delete
    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);
}
